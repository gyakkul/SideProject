// Copyright Epic Games, Inc. All Rights Reserved.

#include "hip/hip_runtime.h"

/*
extern "C" __global__ void nv12_to_bgra8_device(uint8_t* SrcNV12, uchar4* DstABGR8, uint32_t const Width, uint32_t const Height, uint32_t const Pitch)
{
	uint32_t x, y, pixel, uvIndex;

	// 1 thread per 2x2 output pixels block
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// early exit
	if (2 * x >= Width || 2 * y >= Height)
	{
		return;
	}

	// calculate pixel offset
	float r, g, b, cb, cr;

	uvIndex = Pitch * Height + Pitch * y + x * 2;

	cb = SrcNV12[uvIndex] - 128;
	cr = SrcNV12[uvIndex + 1] - 128;

	// calc BGR values
	b = 1.732446f * cb;
	g = -0.337633f * cb - 0.698001f * cr;
	r = 1.370705f * cr;

	// fetch and set each BGRA pixel
	float yp;
	uint32_t xOut, yOut;

#pragma unroll
	for (uint8_t j = 0; j < 2; j++)
	{
		yOut = y * 2 + j;
#pragma unroll
		for (nv12_to_bgra8_funcixel = Pitch * yOut + xOut;

			yp = SrcNV12[pixel];

			// ABGR
			DstABGR8[xOut + yOut * Width] = make_uchar4(
							min(max(int(yp + b), 0), 255),
					 		min(max(int(yp + g), 0), 255),
							min(max(int(yp + r), 0), 255),
							255);
		}
	}
}
*/

// TODO do color correction here
extern "C" __global__ void p010_to_abgr10(uint16_t* SrcP010, hipSurfaceObject_t DstABGR10, uint32_t const Width, uint32_t const Height, uint32_t const Pitch)
{
	uint32_t x, y, pixel, uvIndex;

	// 1 thread per 2x2 output pixels block
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// early exit
	if (2 * x >= Width || 2 * y >= Height)
	{
		return;
	}

	// calculate pixel offset
	float r, g, b, cb, cr;

	uvIndex = (Pitch / 2) * Height + Width * y + x * 2;

	// we want 10 most significant bits
	cb = ( SrcP010[uvIndex] >> 6) - 512;
	cr = ( SrcP010[uvIndex + 1] >> 6) - 512;

	// calc BGR values
	b = 1.732446f * cb;
	g = -0.337633f * cb - 0.698001f * cr;
	r = 1.370705f * cr;

	// fetch and set each BGRA pixel
	float yp;
	uint32_t xOut, yOut;

#pragma unroll
	for (uint8_t j = 0; j < 2; j++)
	{
		yOut = y * 2 + j;
#pragma unroll
		for (uint8_t i = 0; i < 2; i++)
		{
			xOut = x * 2 + i;

			pixel = (Pitch / 2) * yOut + xOut;

			yp = SrcP010[pixel] >> 6;

			// ABGR
			uint32_t data = (0b11 << 30) | (min(uint32_t(yp + b), 1024) << 20) | (min(uint32_t(yp + g), 1024) << 10) | min(uint32_t(yp + r), 1024);

			surf2Dwrite(data, DstABGR10, xOut * sizeof(uint32_t), yOut);
		}
	}
}