// Copyright Epic Games, Inc. All Rights Reserved.

#include "hip/hip_runtime.h"

extern "C" __global__ void nv12_to_bgra8_device(uint8_t* SrcNV12, uchar4* DstABGR8, uint32_t const Width, uint32_t const Height, uint32_t const Pitch)
{
	uint32_t x, y, pixel, uvIndex;

	// 1 thread per 2x2 output pixels block
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// early exit
	if (2 * x >= Width || 2 * y >= Height)
	{
		return;
	}

	// calculate pixel offset
	float r, g, b, cb, cr;

	uvIndex = Pitch * Height + Pitch * y + x * 2;

	cb = SrcNV12[uvIndex] - 128;
	cr = SrcNV12[uvIndex + 1] - 128;

	// calc BGR values
	b = 1.732446f * cb;
	g = -0.337633f * cb - 0.698001f * cr;
	r = 1.370705f * cr;

	// fetch and set each BGRA pixel
	float yp;
	uint32_t xOut, yOut;

#pragma unroll
	for (uint8_t j = 0; j < 2; j++)
	{
		yOut = y * 2 + j;
#pragma unroll
		for (uint8_t i = 0; i < 2; i++)
		{
			xOut = x * 2 + i;

			pixel = Pitch * yOut + xOut;

			yp = SrcNV12[pixel];

			// ABGR
			DstABGR8[xOut + yOut * Width] = make_uchar4(
							min(max(int(yp + b), 0), 255),
					 		min(max(int(yp + g), 0), 255),
							min(max(int(yp + r), 0), 255),
							255);
		}
	}
}

extern "C" __global__ void nv12_to_bgra8(uint8_t* SrcNV12, hipSurfaceObject_t DstABGR8, uint32_t const Width, uint32_t const Height, uint32_t const Pitch)
{
	uint32_t x, y, pixel, uvIndex;

	// 1 thread per 2x2 output pixels block
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// early exit
	if (2 * x >= Width || 2 * y >= Height)
	{
		return;
	}

	// calculate pixel offset
	float r, g, b, cb, cr;

	uvIndex = Pitch * Height + Pitch * y + x * 2;

	cb = SrcNV12[uvIndex] - 128;
	cr = SrcNV12[uvIndex + 1] - 128;

	// calc BGR values
	b = 1.732446f * cb;
	g = -0.337633f * cb - 0.698001f * cr;
	r = 1.370705f * cr;

	// fetch and set each BGRA pixel
	float yp;
	uint32_t xOut, yOut;

#pragma unroll
	for (uint8_t j = 0; j < 2; j++)
	{
		yOut = y * 2 + j;
#pragma unroll
		for (uint8_t i = 0; i < 2; i++)
		{
			xOut = x * 2 + i;

			pixel = Pitch * yOut + xOut;

			yp = SrcNV12[pixel];

			// ABGR
			uchar4 data = { min(max(int(yp + b), 0), 255),
					 		min(max(int(yp + g), 0), 255),
							min(max(int(yp + r), 0), 255),
							255};

			surf2Dwrite(data, DstABGR8, xOut * sizeof(uchar4), yOut);
		}
	}
}

extern "C" __global__ void nv12_to_r8(uint8_t* SrcNV12, hipSurfaceObject_t DstABGR8, uint32_t const Width, uint32_t const Height, uint32_t const Pitch)
{
	uint32_t x, y, pixel, uvIndex;

	// 1 thread per 4 output pixels
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;

	// early exit
	if (2 * x >= Width || 2 * y >= Height)
	{
		return;
	}

	// calculate pixel offset
	float r, g, b, cb, cr;

	uvIndex = Pitch * Height + Pitch * y + x * 2;

	cb = SrcNV12[uvIndex] - 128;
	cr = SrcNV12[uvIndex + 1] - 128;

	// calc BGR values
	b = 2.017 * cb;
	g = -0.392 * cb - 0.813 * cr;
	r = 1.596 * cr;

	// fetch and set each BGRA pixel
	float yp;
	uint32_t xOut, yOut;

#pragma unroll
	for (uint8_t j = 0; j < 2; j++)
	{
		yOut = y * 2 + j;
#pragma unroll
		for (uint8_t i = 0; i < 2; i++)
		{
			xOut = (x * 2 + i) * 4;

			pixel = Pitch * yOut + xOut;

			yp = 1.164 * (SrcNV12[pixel] - 16);

			// ABGR
			unsigned char data;
#pragma unroll
			for (uint8_t c = 0; c < 4; c++)
			{
				switch(c)
				{
					case 0: // B
						data = min(max(int(yp + b), 0), 255);
						break;
					case 1: // G
						data = min(max(int(yp + g), 0), 255);
						break;
					case 2: // R
						data = min(max(int(yp + r), 0), 255);
						break;
					default: // A
						data = 255u;
				}

				surf2Dwrite(data, DstABGR8, (xOut + c) * sizeof(unsigned char), yOut);
			}
		}
	}
}