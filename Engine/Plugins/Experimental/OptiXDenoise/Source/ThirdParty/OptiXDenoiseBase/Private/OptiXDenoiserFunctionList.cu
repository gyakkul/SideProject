#include "hip/hip_runtime.h"
// Copyright Epic Games, Inc. All Rights Reserved.

#include <optix.h>
#include <hip/hip_runtime.h>

struct FFloatReadAccess
{
    inline FFloatReadAccess( const OptixImage2D& InImage )
        : Image( InImage )
        , PixelStrideInBytes( InImage.pixelStrideInBytes ){}

    inline __device__ float read( int x, int y, int c ) const
    {
    	return float( *(const float*)( Image.data + y * Image.rowStrideInBytes + x * PixelStrideInBytes + c * sizeof( float ) ) );
    }
    OptixImage2D Image;
    unsigned int PixelStrideInBytes;
};

struct FFloatWriteAccess
{
    inline FFloatWriteAccess( const OptixImage2D& InImage )
        : Image( InImage )
        , PixelStrideInBytes( InImage.pixelStrideInBytes ){}

    inline __device__ void write( int x, int y, int c, float value )
    {
        *(float*)( Image.data + y * Image.rowStrideInBytes + x * PixelStrideInBytes + c * sizeof( float ) ) = value;
    }
    OptixImage2D Image;
    unsigned int PixelStrideInBytes;
};

static __device__ inline float Dot(float3 A, float3 B)
{
	return A.x * B.x + A.y * B.y + A.z * B.z;
}

static __global__ void k_CopySurfaceToBuffer(
	hipSurfaceObject_t Surface,
	FFloatWriteAccess Result, 
	bool IsNormalInUEViewSpace,
	float PreExposure)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= Result.Image.width || y >= Result.Image.height)
		return;
	
	unsigned int BytesPerPixel = 4 * sizeof(float);
	float4 Data = make_float4(1.0, 0.0,0.0, 1.0);

	surf3Dread(&Data, Surface, x * BytesPerPixel, y, 0);

	if (IsNormalInUEViewSpace)
	{
		// 1. Convert UE space (left handed, y up, view in z) to space used by Optix Denoiser (right handed, y up, view in -z)
		// 2. Normalize from [-1 1] to [0 1]
		const float3 UESpaceToOptiXSpaceMat0 = make_float3(1, 0, 0);
		const float3 UESpaceToOptiXSpaceMat1 = make_float3(0, 1, 0);
		const float3 UESpaceToOptiXSpaceMat2 = make_float3(0, 0,-1);

		float3 Normal = make_float3(Data.x, Data.y, Data.z);
		Data.x = 0.5 * Dot(UESpaceToOptiXSpaceMat0, Normal) + 0.5;
		Data.y = 0.5 * Dot(UESpaceToOptiXSpaceMat1, Normal) + 0.5;
		Data.z = 0.5 * Dot(UESpaceToOptiXSpaceMat2, Normal) + 0.5;
	}

	Result.write(x, y, 0, Data.x * PreExposure);
	Result.write(x, y, 1, Data.y * PreExposure);
	Result.write(x, y, 2, Data.z * PreExposure);
	Result.write(x, y, 3, Data.w * PreExposure);
}

static __global__ void k_CopyBufferToSurface(
	hipSurfaceObject_t Surface, 
	FFloatReadAccess Input)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= Input.Image.width || y >= Input.Image.height)
		return;
	
	float4 Data;
	
	Data.x = Input.read(x, y, 0);
	Data.y = Input.read(x, y, 1);
	Data.z = Input.read(x, y, 2);
	Data.w = Input.read(x, y, 3);

	surf3Dwrite(Data, Surface, x * (4 * sizeof(float)), y, 0);
}

// k_convertRGBA & k_ConvertFlow adapted from optixOpticalFlow.cu for optical flow processing
static __global__ void k_ConvertRGBA(unsigned char* Result, FFloatReadAccess Input, int outStrideX)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= Input.Image.width || y >= Input.Image.height)
		return;

	unsigned int r = __saturatef(Input.read(x, y, 0)) * 255.f;
	unsigned int g = __saturatef(Input.read(x, y, 1)) * 255.f;
	unsigned int b = __saturatef(Input.read(x, y, 2)) * 255.f;

	// OpticalFlow only supports 8bit channel if the input is of rgba.
	*((unsigned int*)&Result[y * outStrideX + x * 4]) = b | (g << 8) | (r << 16) | (255u << 24);
}

static __global__ void k_ConvertFlow(FFloatWriteAccess Result, const int16_t* Input, int inStrideX)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= Result.Image.width || y >= Result.Image.height)
		return;

	// The input data is 2 fixed float (16 bit), with the last 5 bits as fraction.
	Result.write(x, y, 0, float(Input[y * inStrideX + x * 2 + 0]) * (1.f / 32.f));
	Result.write(x, y, 1, float(Input[y * inStrideX + x * 2 + 1]) * (1.f / 32.f));
}